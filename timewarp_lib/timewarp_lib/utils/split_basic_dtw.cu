#include "hip/hip_runtime.h"
#include <torch/extension.h>

__global__
void fill_parent_path_dictionary_cuda_kernel(const torch::PackedTensorAccessor32<float,3> distance_matrix,
                   torch::PackedTensorAccessor32<int,3> parent_path_dictionary,
                   torch::PackedTensorAccessor32<float,3> warping_cost) {
  int64_t recon_time_steps = distance_matrix.size(1);
  int64_t actual_time_steps = distance_matrix.size(2);
  int64_t batch_id = blockIdx.x * blockDim.x + threadIdx.x;

  if (batch_id < distance_matrix.size(0)){

    // for computational simplicity, we assume
    // path costs are 1 for left, diag, and right
    int64_t first_ts = distance_matrix.size(1);
    int64_t second_ts = distance_matrix.size(2);

    // Taking as input
    // ***distance_matrix***
    // Use Dynamic Programming to compute the
    // ***warping_cost*** of a path going to the relevant cell.
    // For ease of subsequent calculation, also keep track
    // of how that cell was reached (eg: diagonally or horizontally)
    // Result is coded in 
    // ***parent_path_dictionary*** using 0,1,2 for
    // parent equal to [(i-1,j-1),(i-1, j),(i][j-1)]
    for (int64_t i = 0; i < first_ts; i++) {
      for (int64_t j = 0; j < second_ts; j++) {
        if (i==0 and j==0) {
          warping_cost[batch_id][i][j] = 0;
        } else if (i == 0) {
          parent_path_dictionary[batch_id][i][j] = 2;
          warping_cost[batch_id][i][j] = warping_cost[batch_id][i][j-1] + distance_matrix[batch_id][i][j];
        } else if (j == 0) {
          parent_path_dictionary[batch_id][i][j] = 1;
          warping_cost[batch_id][i][j] = warping_cost[batch_id][i-1][j] + distance_matrix[batch_id][i][j];
        } 
          // this next part only looks so clean because
          // we're assuming path costs of 1 for left, diag, right.
          // the ordering is to prefer diag over the others if all equal.
          else if (((warping_cost[batch_id][i-1][j-1] <= warping_cost[batch_id][i][j-1]) &&
                    (warping_cost[batch_id][i-1][j-1] <= warping_cost[batch_id][i-1][j]))) {
          parent_path_dictionary[batch_id][i][j] = 0;
          warping_cost[batch_id][i][j] = warping_cost[batch_id][i-1][j-1] + distance_matrix[batch_id][i][j];
        } else if ((warping_cost[batch_id][i-1][j] <= warping_cost[batch_id][i][j-1])) {
          parent_path_dictionary[batch_id][i][j] = 1;
          warping_cost[batch_id][i][j] = warping_cost[batch_id][i-1][j] + distance_matrix[batch_id][i][j];
        } else {
          parent_path_dictionary[batch_id][i][j] = 2;
          warping_cost[batch_id][i][j] = warping_cost[batch_id][i][j-1] + distance_matrix[batch_id][i][j];
        }
      }
    }
  }
}
__global__
void fill_warp_matrices_cuda_kernel(
                   const torch::PackedTensorAccessor32<int,3> parent_path_dictionary,
                   torch::PackedTensorAccessor32<int,3> path,
                   torch::PackedTensorAccessor32<float,3> warp_matrix_first,
                   torch::PackedTensorAccessor32<float,3> warp_matrix_second) {
  int64_t batch_id = blockIdx.x * blockDim.x + threadIdx.x;
  if (batch_id < warp_matrix_first.size(0)){
    int64_t first_ts = parent_path_dictionary.size(1);
    int64_t second_ts = parent_path_dictionary.size(2);
    int64_t maxlen = warp_matrix_first.size(1);

    // using as input the
    // ***parent_path_dictionary***
    // fill in 
    // ***path***
    // backward from end to beginning
    // path ends by definition when it hits (0,0)
    // so, default fill with 0,0s
    int64_t path_len = 0;
    int64_t cur_cell_x = first_ts-1;
    int64_t cur_cell_y = second_ts-1;
    path[batch_id][path_len][0] = cur_cell_x;
    path[batch_id][path_len][1] = cur_cell_y;
    path_len++;
    while (cur_cell_x != 0 or cur_cell_y != 0) {
      int64_t parent_direction = parent_path_dictionary[batch_id][cur_cell_x][cur_cell_y];
      if (parent_direction == 0) {
        cur_cell_x--;
        cur_cell_y--;
      } else if (parent_direction == 1) {
        cur_cell_x--;
      } else {
        cur_cell_y--;
      }

      path[batch_id][path_len][0] = cur_cell_x;
      path[batch_id][path_len][1] = cur_cell_y;
      path_len++;
    }

    // using just the
    // ***path***
    // compute the
    // ***warp_matrices***
    // go through the whole path once, keeping track of when we start
    // and stop matching a particular actual_index
    // NOTE THAT WE go through the path backward (from the last to first)
    // at the beginning, the current_actual_index is the last index
    // For fun, we also fill out the matrix in reverse
    // meaning that our warped path starts with some number of zeros and ends
    // with the path in the right order.
    int64_t current_actual_index = second_ts - 1;
    int64_t start_matching_index = 0;
    int64_t path_index = 0;
    // pair_inds are (recon, actual)
    auto pair_inds = path[batch_id][path_index];
    // (0,0) means you're on your last loop
    bool already_reached_end_of_path = false;
    while (not already_reached_end_of_path) {
      pair_inds = path[batch_id][path_index];
      bool now_at_end_of_path = (pair_inds[0] == 0) && (pair_inds[1] == 0);

      if (pair_inds[1] != current_actual_index) {
        // we've stopped matching the previous match
        // so fill out the previous match (regardless of whether the current
        // cell is the last cell)
        int64_t last_matching_index = path_index - 1;
        int64_t num_matching = last_matching_index - start_matching_index + 1;
        float scaleFactor = sqrt(1./num_matching);
        for (int64_t copy_index = start_matching_index ;
             copy_index <= last_matching_index;
             copy_index++) {
          auto copy_pair = path[batch_id][copy_index];
          // warp_matrix indices are "bar". Not actually smart/useful, 
          // but note how it's different from path_index which is "(r,a)"
          warp_matrix_first[batch_id][maxlen-copy_index-1][copy_pair[0]] =  scaleFactor;
          warp_matrix_second[batch_id][maxlen-copy_index-1][copy_pair[1]] = scaleFactor;
        }
        start_matching_index = path_index;
        current_actual_index = pair_inds[1];
      }
      if (now_at_end_of_path) {
        // The current index is the last correct match if we're now_at_end_of_path
        // Otherwise, the previous index was the last correct match 
        int64_t last_matching_index =  path_index;
        int64_t num_matching = last_matching_index - start_matching_index + 1;
        // we want to weight the _squared_ error by the inverse of number matching...
        // so we need the square root here
        float scaleFactor = sqrt(1./num_matching);
        for (int64_t copy_index = start_matching_index ;
             copy_index <= last_matching_index;
             copy_index++) {
          auto copy_pair = path[batch_id][copy_index];
          // warp_matrix indices are "bar". Not actually smart/useful, 
          // but note how it's different from path_index which is "(r,a)"
          warp_matrix_first[batch_id][maxlen-copy_index-1][copy_pair[0]] =  scaleFactor;
          warp_matrix_second[batch_id][maxlen-copy_index-1][copy_pair[1]] =  scaleFactor;
        }
        start_matching_index = path_index;
        current_actual_index = pair_inds[1];
      }
      path_index++;
      already_reached_end_of_path = now_at_end_of_path;
    }
  }
}

__global__
void fill_warp_matrix_cuda_kernel(
                   const torch::PackedTensorAccessor32<int,3> parent_path_dictionary,
                   torch::PackedTensorAccessor32<int,3> path,
                   torch::PackedTensorAccessor32<float,3> warp_matrix) {
  int64_t batch_id = blockIdx.x * blockDim.x + threadIdx.x;
  if (batch_id < warp_matrix.size(0)){
    // no idea why/when I switched warp_matrix to shape (batch, actual, recon) but too late now
    int64_t first_ts = warp_matrix.size(2);
    int64_t second_ts = warp_matrix.size(1);

    // using as input the
    // ***parent_path_dictionary***
    // fill in 
    // ***path***
    // backward from end to beginning
    // path ends by definition when it hits (0,0)
    // so, default fill with 0,0s
    int64_t path_len = 0;
    int64_t cur_cell_x = first_ts-1;
    int64_t cur_cell_y = second_ts-1;
    path[batch_id][path_len][0] = cur_cell_x;
    path[batch_id][path_len][1] = cur_cell_y;
    path_len++;
    while (cur_cell_x != 0 or cur_cell_y != 0) {
      int64_t parent_direction = parent_path_dictionary[batch_id][cur_cell_x][cur_cell_y];
      if (parent_direction == 0) {
        cur_cell_x--;
        cur_cell_y--;
      } else if (parent_direction == 1) {
        cur_cell_x--;
      } else {
        cur_cell_y--;
      }

      path[batch_id][path_len][0] = cur_cell_x;
      path[batch_id][path_len][1] = cur_cell_y;
      path_len++;
    }

    // using just the
    // ***path***
    // compute the
    // ***warp_matrix***
    // To do so, go through the whole path once, keeping track of when we start
    // and stop matching a particular actual_index
    // since we go through the path backward (from the last to first)
    // at the beginning, the current_actual_index is the last index
    int64_t current_actual_index = second_ts-1;
    int64_t start_matching_index= 0;
    int64_t path_index = 0;
    // pair_inds are (recon, actual)
    auto pair_inds = path[batch_id][path_index];
    // (0,0) means you're on your last loop
    bool already_reached_end_of_path = false;
    while (not already_reached_end_of_path) {
      pair_inds = path[batch_id][path_index];
      bool now_at_end_of_path = (pair_inds[0] == 0) && (pair_inds[1] == 0);

      if (pair_inds[1] != current_actual_index) {
        // we've stopped matching the previous match
        // so fill out the previous match (regardless of whether the current
        // cell is the last cell)
        int64_t last_matching_index = path_index - 1;
        int64_t num_matching = last_matching_index - start_matching_index + 1;
        for (int64_t copy_index = start_matching_index ;
             copy_index <= last_matching_index;
             copy_index++) {
          auto copy_pair = path[batch_id][copy_index];
          // warp_matrix indices are "bar". Not actually smart/useful, 
          // but note how it's different from path_index which is "(r,a)"
          warp_matrix[batch_id][copy_pair[1]][copy_pair[0]] =  1./num_matching;
        }
        start_matching_index = path_index;
        current_actual_index = pair_inds[1];
      }
      if (now_at_end_of_path) {
        // The current index is the last correct match if we're now_at_end_of_path
        // Otherwise, the previous index was the last correct match 
        int64_t last_matching_index =  path_index;
        int64_t num_matching = last_matching_index - start_matching_index + 1;
        for (int64_t copy_index = start_matching_index ;
             copy_index <= last_matching_index;
             copy_index++) {
          auto copy_pair = path[batch_id][copy_index];
          // warp_matrix indices are "bar". Not actually smart/useful, 
          // but note how it's different from path_index which is "(r,a)"
          warp_matrix[batch_id][copy_pair[1]][copy_pair[0]] =  1./num_matching;
        }
        start_matching_index = path_index;
        current_actual_index = pair_inds[1];
      }
      path_index++;
      already_reached_end_of_path = now_at_end_of_path;
    }
  }
}

// given a matrix of shape (batch_size, recon_time_steps, actual_time_steps)
// giving the distance between recon and actual
// compute the dtw path that aligns the recon and actual
// with one output:
// path_base of shape (batch_size, recon_time_steps + actual_time_steps + 1, 2)
// giving pairs (recon_time, actual_time) along the path
// path_base is an "out" parameter that gets filled in
void fill_parent_path_dict(torch::Tensor distance_matrix_base, torch::Tensor& parent_path_dictionary_base) {
  int64_t batch_size = distance_matrix_base.size(0);
  int64_t recon_time_steps = distance_matrix_base.size(1);
  int64_t actual_time_steps = distance_matrix_base.size(2);

  auto warp_options = torch::TensorOptions().device(torch::kCUDA);

  auto parent_path_dictionary = parent_path_dictionary_base.packed_accessor32<int,3>();

  const int threads = 1024;
  const int blocks = (batch_size + threads - 1) / threads;

  auto distance_matrix = distance_matrix_base.packed_accessor32<float,3>();

  auto warping_cost_base = torch::zeros_like(distance_matrix_base,warp_options);
  auto warping_cost = warping_cost_base.packed_accessor32<float,3>();

  AT_DISPATCH_FLOATING_TYPES(distance_matrix_base.type(), "fill_parent_path_dictionary_dtw_cuda", ([&] {
    fill_parent_path_dictionary_cuda_kernel<<<blocks,threads>>>(distance_matrix, parent_path_dictionary, warping_cost);
  }));
}

// given a matrix of parent_path_dictionary
// compute the warping matrix you can use to convert the recon data to align it
// to the actual data
// warp_matrix_base is an "out" parameter that gets filled in
void fill_warp_recon_to_actual(torch::Tensor parent_path_dictionary_base, torch::Tensor& warp_matrix_base) {
  int64_t batch_size = parent_path_dictionary_base.size(0);
  int64_t recon_time_steps = parent_path_dictionary_base.size(1);
  int64_t actual_time_steps = parent_path_dictionary_base.size(2);
  auto path_options = torch::TensorOptions().dtype(torch::kInt32).device(torch::kCUDA);
  auto path_base = torch::zeros({batch_size,recon_time_steps+actual_time_steps+1,2},path_options);
  auto path = path_base.packed_accessor32<int,3>();
  auto warp_matrix = warp_matrix_base.packed_accessor32<float,3>();
  auto parent_path_dictionary = parent_path_dictionary_base.packed_accessor32<int,3>();
  
  const int threads = 1024;
  const int blocks = (batch_size + threads - 1) / threads;

  AT_DISPATCH_FLOATING_TYPES(warp_matrix_base.type(), "fill_warp_matrix_dtw_cuda", ([&] {
    fill_warp_matrix_cuda_kernel<<<blocks,threads>>>(parent_path_dictionary, path, warp_matrix);
  }));
}

// given a matrix of parent_path_dictionary
// compute the warping matrices you can use to 
// convert BOTH the recon data AND the actual data to align them
// warp_matrix_base is an "out" parameter that gets filled in
void fill_warp_recon_and_actual(torch::Tensor parent_path_dictionary_base, torch::Tensor& warp_matrix_first_base, torch::Tensor& warp_matrix_second_base) {
  int64_t batch_size = parent_path_dictionary_base.size(0);
  int64_t recon_time_steps = parent_path_dictionary_base.size(1);
  int64_t actual_time_steps = parent_path_dictionary_base.size(2);
  auto path_options = torch::TensorOptions().dtype(torch::kInt32).device(torch::kCUDA);
  auto path_base = torch::zeros({batch_size,recon_time_steps+actual_time_steps+1,2},path_options);
  auto path = path_base.packed_accessor32<int,3>();
  auto warp_matrix_first = warp_matrix_first_base.packed_accessor32<float,3>();
  auto warp_matrix_second= warp_matrix_second_base.packed_accessor32<float,3>();
  auto parent_path_dictionary = parent_path_dictionary_base.packed_accessor32<int,3>();
  
  const int threads = 1024;
  const int blocks = (batch_size + threads - 1) / threads;

  AT_DISPATCH_FLOATING_TYPES(warp_matrix_first_base.type(), "fill_warp_matrix_dtw_cuda", ([&] {
    fill_warp_matrices_cuda_kernel<<<blocks,threads>>>(parent_path_dictionary, path, warp_matrix_first, warp_matrix_second);
  }));
}
